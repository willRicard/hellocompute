
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <math.h>

__global__ void add(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    y[i] = x[i] + y[i];
  }
}

int main(int argc, char **argv) {
  int N = 1 << 20;
  float *x = nullptr, *y = nullptr;

  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  for (int i = 0; i < N; ++i) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  auto start = std::chrono::high_resolution_clock::now();

  int numBlocks = (N + 256 - 1) / 256;
  add<<<numBlocks, 256>>>(N, x, y);

  auto end = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);
  std::cout << 1000.0 * duration.count() << " ms" << std::endl;

  hipDeviceSynchronize();

  float maxError = 0.0f;
  for (int i = 0; i < N; ++i) {
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  }
  std::cout << "Max error: " << maxError << std::endl;

  hipFree(x);
  hipFree(y);

  return 0;
}
